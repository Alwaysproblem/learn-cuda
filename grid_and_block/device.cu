#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cstdio>
#include "device.h"

__global__ void checkIndex_imp(void)
{
    printf("threadIdx:(%d, %d, %d)\n", threadIdx.x, threadIdx.y, threadIdx.z);
    printf("blockIdx:(%d, %d, %d)\n", blockIdx.x, blockIdx.y, blockIdx.z);
    printf("--------------------------------\n");
    printf("blockDim:(%d, %d, %d)\n", blockDim.x, blockDim.y, blockDim.z);
    printf("gridDim:(%d, %d, %d)\n", gridDim.x, gridDim.y, gridDim.z);
}

// 在 device.cu 中封装函数调用
extern "C" void checkIndex(dim3 grid, dim3 block) {
    checkIndex_imp<<<grid, block>>>();
    hipDeviceReset();
}
