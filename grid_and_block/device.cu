#include "hip/hip_runtime.h"
#include <stdio.h>

#include <cstdio>

#include "device.h"

__global__ void checkIndex_imp(void) {
  printf(
      "threadIdx:(%d, %d, %d) blockIdx:(%d, %d, %d) blockDim:(%d, %d, %d) "
      "gridDim:(%d, %d, %d)\n",
      threadIdx.x, threadIdx.y, threadIdx.z, blockIdx.x, blockIdx.y, blockIdx.z,
      blockDim.x, blockDim.y, blockDim.z, gridDim.x, gridDim.y, gridDim.z);
}

// 在 device.cu 中封装函数调用
extern "C" void checkIndex(dim3 grid, dim3 block) {
  checkIndex_imp<<<grid, block>>>();
  hipDeviceReset();
}
