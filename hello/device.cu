#include "hip/hip_runtime.h"
/*
 * A simple introduction to programming in CUDA. This program prints "Hello
 * World from GPU! from 10 CUDA threads running on the GPU.
 */

#include "device.h"

__global__ void helloFromGPU()
{
    printf("Hello World from GPU!\n");
}

// 在 device.cu 中封装函数调用
extern "C" void launch_helloFromGPU() {
    helloFromGPU<<<1, 10>>>();
    hipDeviceSynchronize();
}
