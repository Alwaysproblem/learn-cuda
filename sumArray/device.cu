#include "hip/hip_runtime.h"
#include "device.h"

__global__ void sumArray(float *A, float *B, float *C, const int N) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < N) {
    C[i] = A[i] + B[i];
  }
}

// 在 device.cu 中封装函数调用
extern "C" void sumArrayOnGPU(float *A, float *B, float *C, const int N,
                              const int grid_size, const int block_size) {
  dim3 block(grid_size);
  dim3 grid(block_size);
  float *d_A, *d_B, *d_C;
  hipMalloc((float **)&d_A, N * sizeof(float));
  hipMalloc((float **)&d_B, N * sizeof(float));
  hipMalloc((float **)&d_C, N * sizeof(float));
  hipMemcpy(d_A, A, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, N * sizeof(float), hipMemcpyHostToDevice);
  sumArray<<<block, grid>>>(d_A, d_B, d_C, N);
  hipMemcpy(C, d_C, N * sizeof(float), hipMemcpyDeviceToHost);
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
}
