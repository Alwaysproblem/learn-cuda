#include "hip/hip_runtime.h"
#include "device.h"

__global__ void sumMatrix(float *A, float *B, float *C, const int nx, const int ny) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  unsigned int idx = i * ny + j;
  if (i < nx && j < ny) {
    C[idx] = A[idx] + B[idx];
  }
}

// 在 device.cu 中封装函数调用
extern "C" void sumMatrixOnGPU(float *A, float *B, float *C, const int nx,
                               const int ny, const dim3 grid,
                               const dim3 block) {
  int N = nx * ny;
  float *d_A, *d_B, *d_C;
  hipMalloc((float **)&d_A, N * sizeof(float));
  hipMalloc((float **)&d_B, N * sizeof(float));
  hipMalloc((float **)&d_C, N * sizeof(float));
  hipMemcpy(d_A, A, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, N * sizeof(float), hipMemcpyHostToDevice);
  sumMatrix<<<block, grid>>>(d_A, d_B, d_C, nx, ny);
  hipMemcpy(C, d_C, N * sizeof(float), hipMemcpyDeviceToHost);
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
}
