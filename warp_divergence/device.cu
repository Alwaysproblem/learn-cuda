#include "hip/hip_runtime.h"
#include "device.h"

#define MATHKERNEL(name)                                                       \
  extern "C" void name(float *c, int N, const dim3 grid, const dim3 block) {   \
    float *d_C;                                                                \
    hipMalloc((float **)&d_C, N * sizeof(float));                             \
    name##OnGPU<<<block, grid>>>(d_C);                                         \
    hipMemcpy(c, d_C, N * sizeof(float), hipMemcpyDeviceToHost);             \
    hipFree(d_C);                                                             \
  }

__global__ void mathKernel1OnGPU(float *c) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  float ia, ib;
  ia = ib = 0.0f;

  if (tid % 2 == 0) {
    ia = 100.0f;
  } else {
    ib = 200.0f;
  }

  c[tid] = ia + ib;
}

__global__ void mathKernel2OnGPU(float *c) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  float ia, ib;
  ia = ib = 0.0f;

  if ((tid / warpSize) % 2 == 0) {
    ia = 100.0f;
  } else {
    ib = 200.0f;
  }

  c[tid] = ia + ib;
}

__global__ void mathKernel3OnGPU(float *c) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  float ia, ib;
  ia = ib = 0.0f;

  bool ipred = (tid % 2 == 0);

  if (ipred) {
    ia = 100.0f;
  }

  if (!ipred) {
    ib = 200.0f;
  }

  c[tid] = ia + ib;
}

__global__ void mathKernel4OnGPU(float *c) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  float ia, ib;
  ia = ib = 0.0f;

  int itid = tid >> 5;

  if (itid & 0x01 == 0) {
    ia = 100.0f;
  } else {
    ib = 200.0f;
  }

  c[tid] = ia + ib;
}

__global__ void warmingupOnGPU(float *c) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  float ia, ib;
  ia = ib = 0.0f;

  if ((tid / warpSize) % 2 == 0) {
    ia = 100.0f;
  } else {
    ib = 200.0f;
  }

  c[tid] = ia + ib;
}

MATHKERNEL(mathKernel1);
MATHKERNEL(mathKernel2);
MATHKERNEL(mathKernel3);
MATHKERNEL(mathKernel4);
MATHKERNEL(warmingup);
